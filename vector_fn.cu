
#include <hip/hip_runtime.h>
#include <iostream>

__global__ void vector_fn(int n, int* vec, int* res)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;
    
    int temp_sum = 0;
    for (int i = idx; i < n; i += stride) {
      temp_sum = temp_sum + vec[i];
    }
    atomicAdd(res, temp_sum);
}

int main() {
    int N = 1024;
    int* vec;
    int* res;
    hipMallocManaged(&vec, N * sizeof(int));
    hipMallocManaged(&res, sizeof(int));
    std::cout << "Done allocation, about to initialize with N=" << N << std::endl;

    *res = 0;
    int true_res = 0;
    for (int i = 0; i < N; i++) {
      vec[i] = i;
      true_res += i;
    }
    
    std::cout << "Done initialization, about to run kernels, expected result = " << true_res << std::endl;
    int blockSize = 32;
    int numBlocks = (N + blockSize - 1) / blockSize;
    vector_fn<<<numBlocks, blockSize>>>(N, vec, res);
    hipDeviceSynchronize();

    std::cout << "vector_fn result: " << *res << std::endl;
    std::cout << "Equal to expected: " << ((true_res==*res) ? "true" : "false") << std::endl;

    hipFree(res);
    hipFree(vec);

    return 0;
}
