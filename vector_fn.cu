
#include <hip/hip_runtime.h>
#include <iostream>

__global__ void vector_fn(int n, int* vec, int* res)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;
    
    int temp_sum = 0;
    for (int i = idx; i < n; i += stride) {
      temp_sum = temp_sum + vec[i];
    }
    atomicAdd(res, temp_sum);
}

// same as above, but with sequential accesses for locality
__global__ void vector_fn_seq(int n, int* vec, int* res)
{
  // gridDim: # of blocks in a grid
  // blockDim: # of threads in a block

    int tid = threadIdx.x;
    int elements_per_block = (n + gridDim.x - 1) / gridDim.x;
    int block_start = blockIdx.x * elements_per_block;
    // int block_end = min(block_start + elements_per_block, n);
    int block_end = block_start + elements_per_block;
    
    int elements_per_thread = (block_end - block_start + blockDim.x - 1) / blockDim.x;
    int thread_start = block_start + tid * elements_per_thread;
    // int thread_end = min(thread_start + elements_per_thread, block_end);
    int thread_end = thread_start + elements_per_thread;
    
    // Sequential access with stride 1
    int temp_sum = 0;
    for (int i = thread_start; i < thread_end; i++) {
        temp_sum += vec[i];
    }
    atomicAdd(res, temp_sum);
}

int main() {
    int N = 1024;
    int* vec;
    int* res;
    hipMallocManaged(&vec, N * sizeof(int));
    hipMallocManaged(&res, sizeof(int));
    std::cout << "done alloc, about to initialize with N=" << N << std::endl;

    int true_res = 0;
    for (int i = 0; i < N; i++) {
      vec[i] = i;
      true_res += i;
    }
    int blockSize = 32;   // num threads per block
    int numBlocks = (N + blockSize - 1) / blockSize;
    
    *res = 0;
    std::cout << "done init, about to run vector_fn kernel, expected result = " << true_res << std::endl;
    vector_fn<<<numBlocks, blockSize>>>(N, vec, res);
    hipDeviceSynchronize();

    *res = 0;
    std::cout << "done init, about to run vector_fn_seq kernel, expected result = " << true_res << std::endl;
    vector_fn_seq<<<numBlocks, blockSize>>>(N, vec, res);
    hipDeviceSynchronize();

    std::cout << "vector_fn result: " << *res << std::endl;
    std::cout << "  -  Equal to expected: " << ((true_res==*res) ? "true" : "false") << std::endl;
    std::cout << "vector_fn_seq result: " << *res << std::endl;
    std::cout << "  -  Equal to expected: " << ((true_res==*res) ? "true" : "false") << std::endl;

    hipFree(res);
    hipFree(vec);

    return 0;
}
